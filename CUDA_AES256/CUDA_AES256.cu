#include "CUDA_AES256.cuh"

namespace cuda_aes {
    hipError_t cuda_init(int device) {
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(device);
        if (cudaStatus != hipSuccess) throw std::runtime_error("Runtime Error | cuda_init(): Error while setting device");

        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(aes::cuda_con::sbox), aes::con::sbox, sizeof(aes::con::sbox));
        if (cudaStatus != hipSuccess) throw std::runtime_error("Runtime Error | cuda_init(): Error copying sbox");

        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(aes::cuda_con::inverse_sbox), aes::con::inverse_sbox, sizeof(aes::con::inverse_sbox));
        if (cudaStatus != hipSuccess) throw std::runtime_error("Runtime Error | cuda_init(): Error copying inverse_sbox");

        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(aes::cuda_con::mixColumnLookup), aes::con::mixColumnLookup, sizeof(aes::con::mixColumnLookup));
        if (cudaStatus != hipSuccess) throw std::runtime_error("Runtime Error | cuda_init(): Error copying mixColumnLookup");

        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(aes::cuda_con::invMixColumnLookup), aes::con::invMixColumnLookup, sizeof(aes::con::invMixColumnLookup));
        if (cudaStatus != hipSuccess) throw std::runtime_error("Runtime Error | cuda_init(): Error copying invMixColumnLookup");
    }
}